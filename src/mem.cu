#include "mem.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define LAYER_MAX 128

CORAMem::CORAMem(size_t total_size, int world_size, int dev_type) {
  // allocate memory for each party
  assert(world_size <= 4 & world_size > 0);
  this->world_size = world_size;
  assert(total_size > 0);
  assert(dev_type == DEV_CPU || dev_type == DEV_GPU);
  this->dev_type = dev_type;

  snd_raw = sys_malloc_internal(total_size);
  this->total_size = total_size;
  this->snd_head = &this->snd_mem[0];
  snd_head->next = nullptr;
  snd_head->size = total_size;
  snd_head->is_free = 1;
  snd_head->ptr = snd_raw;
  snd_head->in_use = 1;


  for (int i = 0; i < world_size; i++) {
    rcv_head[i] = &rcv_mem[i][0];
    rcv_raw[i] = sys_malloc_internal(total_size);
    rcv_head[i]->ptr = rcv_raw[i];
    rcv_head[i]->size = total_size;
    rcv_head[i]->is_free = 1;
    rcv_head[i]->next = nullptr;
    rcv_head[i]->in_use = 1;
  }
  // reserve 128 layers
  layer_mem_map.resize(LAYER_MAX);

}

void* CORAMem::sys_malloc_internal(size_t size) {
  if (dev_type == DEV_CPU) {
    return malloc(size);
  } else {
    void *devptr;
    hipMalloc(&devptr, size);
    return devptr;
  }
}

CORAMem::~CORAMem() {
  if (dev_type == DEV_CPU) {
    free(snd_raw);
    for (int i = 0; i < world_size; i++) {
      free(rcv_raw[i]);
    }
  } else {
    hipFree(snd_raw);
    for (int i = 0; i < world_size; i++) {
      hipFree(rcv_raw[i]);
    }
  }
}

MemBlock* CORAMem::allocate_internal(size_t size) {
  MemBlock* current = snd_head;
  while (current) {
    if (current->is_free && current->size >= size) {
        current->is_free = 0;
        if (current->size == size) { // no need to split block
            return current;
        }
        // split and create new block
        split_block(current, size);
        return current;
    }
    current = current->next;
  }
  return nullptr;
}

MemBlock* CORAMem::find_available_block(MemBlock* block_array, int size) {
  MemBlock* current = block_array;
  for (uint i = 0; i < size; i++) {
    if (current->in_use == 0) {
      return current;
    }
    current++;
  }
  return nullptr;
}

void CORAMem::split_block(MemBlock* block, size_t size) {
  if(block->size <= size) {
    return; // no need to split
  }
  MemBlock* new_block = find_available_block(this->snd_mem, 128);
  assert(new_block != nullptr);
  new_block->in_use = 1;
  new_block->size = block->size - size;
  new_block->is_free = 1;
  new_block->ptr = (char*)block->ptr + size;
  block->size = size;
  new_block->next = block->next;
  block->next = new_block;
}

void CORAMem::merge_block(MemBlock* block1, MemBlock* block2) {
  assert(block1->in_use && block2->in_use);
  assert(block1->is_free && block2->is_free);
  assert(block1->next == block2);
  block1->size += block2->size;
  block1->next = block2->next;
  block2->in_use = 0;
}

void CORAMem::free_internal(MemBlock* block_ptr) {
  block_ptr->is_free = 1;
  // merge free blocks
  MemBlock* cur_block = snd_head;
  MemBlock* prev_block = nullptr;
  // merge prev free blocks
  // first, find the prev block if available
  while(cur_block != nullptr) {
    if (cur_block->next == block_ptr) {
      prev_block = cur_block;
      break;
    }
    cur_block = cur_block->next;
  }

  if (prev_block != nullptr && prev_block->is_free) {
    cur_block = prev_block;
    merge_block(cur_block, block_ptr);
    while (cur_block->next && cur_block->next->is_free) {
      MemBlock* next_block = cur_block->next;
      merge_block(cur_block, next_block);
      cur_block = cur_block->next;
    }
  } else {
    cur_block = block_ptr;
    while(cur_block->next && cur_block->next->is_free) {
      MemBlock* next_block = cur_block->next;
      merge_block(block_ptr, cur_block);
      cur_block = cur_block->next;
    }
  }

}

SymMem CORAMem::Allocate(size_t size, int layer_id) {
  if (layer_id == 0 || layer_id == 1 || layer_id == 2) {
    MemBlock* ptr = allocate_internal(size);
    if (ptr) {
      layer_mem_map[layer_id].push_back(ptr);
      uint64_t offset = (char*)ptr->ptr - (char*)snd_raw;
      return SymMem {
        .local_ptr = ptr->ptr,
        .rem_ptr = {
          rcv_raw[0] + offset,
          rcv_raw[1] + offset,
          rcv_raw[2] + offset,
          rcv_raw[3] + offset
        },
        .size = size
      };
    } else {
      return SymMem {
        .local_ptr = nullptr,
        .rem_ptr = {nullptr, nullptr, nullptr, nullptr},
        .size = 0
      };
    }
  }
  else { // layer_id >= 3
    // free layer_id - 2
    for (auto block : layer_mem_map[layer_id - 2])
      free_internal(block);
    MemBlock* ptr = allocate_internal(size);
    if (ptr) {
      layer_mem_map[layer_id].push_back(ptr);
      uint64_t offset = (char*)ptr->ptr - (char*)snd_raw;
      return SymMem{
        .local_ptr = ptr->ptr,
        .rem_ptr = {
          rcv_raw[0] + offset,
          rcv_raw[1] + offset,
          rcv_raw[2] + offset,
          rcv_raw[3] + offset
        },
        .size = size
      };
    } else {
      return SymMem {
        .local_ptr = nullptr,
        .rem_ptr = {nullptr, nullptr, nullptr, nullptr},
        .size = 0
      };
    }
  }
}


void* CORAMem::GetLocal() {
  return snd_raw;
}

void* CORAMem::GetRemote(int party_id) {
  return rcv_raw[party_id];
}

void CORAMem::PrintLayout() {
  MemBlock* current = snd_head;
  while (current) {
    printf("block ptr: %p, size: %lu, is_free: %d\n", current->ptr, current->size, current->is_free);
    current = current->next;
  }
}
